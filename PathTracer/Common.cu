#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>

#include "Payloads.h"

using namespace optix;

rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(float3, backgroundColor, , );
rtDeclareVariable(int, hasEnvmap, , );
rtTextureSampler<float4, 2> envmap;

RT_PROGRAM void miss()
{
    if (hasEnvmap) {
        float theta = atan2f(payload.dir.x, payload.dir.z);
        float phi = M_PIf * 0.5f - acosf(payload.dir.y);
        float u = (theta + M_PIf) * (0.5f * M_1_PIf);
        float v = 0.5f * (1.0f + sin(phi));
        float3 result = make_float3(tex2D(envmap, u, v));
        // Set the result to be the background color if miss
        payload.radiance = result;
    }
    else {
        payload.radiance = backgroundColor;
    }
    payload.done = true;
}

RT_PROGRAM void exception()
{
    // Print any exception for debugging
    const unsigned int code = rtGetExceptionCode();
    rtPrintExceptionDetails();
}

rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );

RT_PROGRAM void anyHit()
{
    shadowPayload.isVisible = false;
    rtTerminateRay();
}